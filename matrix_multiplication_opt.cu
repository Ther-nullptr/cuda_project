#include "hip/hip_runtime.h"

#include <stdio.h>
#include "matrix_multiplication.h"
#include "util.h"

// Thread Block size
#define BLOCK_SIZE 32


__global__ void MatMulKernel(Matrix A, Matrix B, Matrix C);

// Get a matrix element 
__device__ int GetElement(const Matrix A, int row, int col) {
    return A.elements[row * A.stride + col]; 
} 

// Set a matrix element 
__device__ void SetElement(Matrix A, int row, int col, int value) { 
    A.elements[row * A.stride + col] = value; 
} 

// Get the BLOCK_SIZE x BLOCK_SIZE sub-matrix Asub of A that is 
// located at the col th sub-matrices to the right and the row th sub-matrices down 
// from the upper-left corner of A 
// NOTE that the GetSubMatrix function returns a pointer of Matrix A
__device__ Matrix GetSubMatrix(Matrix A, int row, int col) { 
    Matrix Asub; 
    Asub.width = BLOCK_SIZE; 
    Asub.height = BLOCK_SIZE; 
    Asub.stride = A.stride; 
    Asub.elements = &A.elements[A.stride * BLOCK_SIZE * row + BLOCK_SIZE * col]; 
    return Asub; 
}

void MatMul(const Matrix A, const Matrix B, Matrix C)
{
    ////////////////////////////////
    // CUDA Event Create to estimate elased time
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    // CUDA Operation
    hipEventRecord(start, 0);
    /////////////////////////////////
    
    // Load A and B to device memory
    Matrix d_A;
    d_A.width = d_A.stride = A.width; d_A.height = A.height;
    size_t size = A.width * A.height * sizeof(int);
    hipMalloc(&d_A.elements, size);
    hipMemcpy(d_A.elements, A.elements, size,
               hipMemcpyHostToDevice);

    Matrix d_B;
    d_B.width = d_B.stride = B.width; d_B.height = B.height;
    size = B.width * B.height * sizeof(int);
    hipMalloc(&d_B.elements, size);
    hipMemcpy(d_B.elements, B.elements, size,
    hipMemcpyHostToDevice);

    // Allocate C in device memory
    Matrix d_C;
    d_C.width = d_C.stride = C.width; d_C.height = C.height;
    size = C.width * C.height * sizeof(int);
    hipMalloc(&d_C.elements, size);

    // Launch CUDA Kernel 
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 dimGrid(B.width / dimBlock.x, A.height / dimBlock.y);
    MatMulKernel<<<dimGrid, dimBlock>>>(d_A, d_B, d_C);

    // Read C from device memory
    hipMemcpy(C.elements, d_C.elements, size,
               hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_A.elements);
    hipFree(d_B.elements);
    hipFree(d_C.elements);
    
    /////////////////////////////////
    // Estimate CUDA operation time
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);
    printf("CUDA Elapsed time: %f ms\n", elapsedTime);
    
    // finalize CUDA event
    hipEventDestroy(start);
    hipEventDestroy(stop);
    
}


// Matrix multiplication kernel called by MatMul()
 __global__ void MatMulKernel(Matrix A, Matrix B, Matrix C)
{
    // Block row and column
    int blockRow = blockIdx.y;
    int blockCol = blockIdx.x;

    // Each thread block computes one sub-matrix Csub of C
    Matrix Csub = GetSubMatrix(C, blockRow, blockCol);

    // Each thread computes ONE element of Csub
    // by accumulating results into Cvalue
    int Cvalue = 0;

    // Thread row and column within Csub
    int row = threadIdx.y;
    int col = threadIdx.x;

    // Loop over all the sub-matrices of A and B that are
    // required to compute Csub
    // Multiply each pair of sub-matrices together
    // and accumulate the results
    for (int m = 0; m < (A.width / BLOCK_SIZE); m++) {

        // Use GetSubMatrix function to get sub-matrix Asub of A
        /*******************TODO*******************/

        Matrix Asub = GetSubMatrix(A, blockRow, m);

        // Use GetSubMatrix function to get sub-matrix Bsub of B
        /*******************TODO*******************/ 

        Matrix Bsub = GetSubMatrix(B, m, blockCol);

        // Shared memory used to store Asub and Bsub respectively
        __shared__ int As[BLOCK_SIZE][BLOCK_SIZE];
        __shared__ int Bs[BLOCK_SIZE][BLOCK_SIZE];

        // Load Asub and Bsub from device memory to shared memory
        // Each thread loads one element of each sub-matrix. 
        // You can use GetElement function.
        /*******************TODO*******************/

        As[row][col] = GetElement(Asub, row, col);
        Bs[row][col] = GetElement(Bsub, row, col);

        // Synchronize to make sure the sub-matrices are loaded
        // before starting the computation
        __syncthreads();
        // Multiply Asub and Bsub together
        // Each thread computes ONE element of Csub
        // by accumulating results into Cvalue
        /*******************TODO*******************/

        for (int e = 0; e < BLOCK_SIZE; ++e)
        {
            Cvalue += As[row][e] * Bs[e][col];
        }

        // Synchronize to make sure that the preceding
        // computation is done before loading two new
        // sub-matrices of A and B in the next iteration
        __syncthreads();
    }

    // Write Csub to device memory
    // Each thread writes one element
    // You can use SetElement function.
    /*******************TODO*******************/
    SetElement(Csub, row, col, Cvalue);
}

int main() {
    int width_a = 256*4;
    int height_a = 128*4;
    int height_b = 256*4;
    int width_b = height_a;
    
    Matrix A, B, C, C_cuda;
    
    init_matrix(&A, width_a, height_a, 1);
    init_matrix(&B, width_b, height_b, 2);
    init_matrix(&C, A.height, B.width, 0);
    init_matrix(&C_cuda, A.height, B.width, 0);
    
    // Matrix Multiplication
    MatMul(A, B, C_cuda);
    matrix_multiplication_host(C, A, B);
    
    // Check results
    check_result(C, C_cuda);
    
    free(A.elements);
    free(B.elements);
    free(C.elements);
    free(C_cuda.elements);
    
    return 0;
}