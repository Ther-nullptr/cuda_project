#include "hip/hip_runtime.h"

#include <stdio.h>
#include "matrix_multiplication.h"
#include "util.h"

__global__
void d_matrix_multiplication(Matrix C, Matrix A, Matrix B) {

    
    int idx_x = blockDim.x * blockIdx.x + threadIdx.x;
    int idx_y = blockDim.y * blockIdx.y + threadIdx.y;

	//Note: use A.elements[] to get the elements from A. So does B. 
	//You can see the definitions in matrix_multiplication.h
	/*******************TODO*******************/

    int sum = 0;
    for (int i = 0; i < A.width; i++) {
        sum += A.elements[idx_y * A.width + i] * B.elements[i * B.width + idx_x];
    }
    C.elements[idx_y * C.width + idx_x] = sum;
}

void matrix_multiplication(Matrix &C, Matrix A, Matrix B) {
    int size;
    
    ////////////////////////////////
    // CUDA Event Create to estimate elased time
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    // CUDA Operation
    hipEventRecord(start, 0);
    /////////////////////////////////
    
    // Create GPU memory
    Matrix d_A;
    d_A.width = A.width; d_A.height = A.height;
    size = A.width * A.height * sizeof(int);
    hipMalloc((void**)&d_A.elements, size);
    hipMemcpy(d_A.elements, A.elements, size, hipMemcpyHostToDevice);
    
    Matrix d_B;
    d_B.width = B.width; d_B.height = B.height;
    size = B.width * B.height * sizeof(int);
    hipMalloc((void**)&d_B.elements, size);
    hipMemcpy(d_B.elements, B.elements, size, hipMemcpyHostToDevice);
    
    Matrix d_C;
    d_C.height = C.height; d_C.width = C.width;
    size = C.width * C.height * sizeof(int);
    hipMalloc((void**)&d_C.elements, size);
    
    // Launch CUDA Kernel 
    dim3 blockDim(8, 8);  
    dim3 gridDim(d_C.width / blockDim.x, d_C.height / blockDim.y);

    printf("blockDim: %d, %d\n", blockDim.x, blockDim.y);
    printf("gridDim: %d, %d\n", gridDim.x, gridDim.y);

    d_matrix_multiplication<<<gridDim, blockDim>>>(d_C, d_A, d_B);

    hipMemcpy(C.elements, d_C.elements, size, hipMemcpyDeviceToHost);
    
    hipFree(d_A.elements);
    hipFree(d_B.elements);
    hipFree(d_C.elements);
    
    /////////////////////////////////
    // Estimate CUDA operation time
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);
    printf("CUDA Elapsed time: %f ms\n", elapsedTime);
    
    // finalize CUDA event
    hipEventDestroy(start);
    hipEventDestroy(stop);
    /////////////////////////////////
}

int main() {
    int width_a = 256*4;
    int height_a = 128*4;
    int height_b = 256*4;
    int width_b = height_a;
    
    Matrix A, B, C, C_cuda;
    
    init_matrix(&A, width_a, height_a, 1);
    init_matrix(&B, width_b, height_b, 2);
    init_matrix(&C, A.height, B.width, 0);
    init_matrix(&C_cuda, A.height, B.width, 0);
    
    // Matrix Multiplication
    matrix_multiplication(C_cuda, A, B);
    matrix_multiplication_host(C, A, B);
    
    // Check results
    check_result(C, C_cuda);
    
    free(A.elements);
    free(B.elements);
    free(C.elements);
    free(C_cuda.elements);
     
    return 0;
}